#include "hip/hip_runtime.h"
#include <stdio.h>           
#include <stdlib.h>        
#include <hip/hip_runtime.h>
#include ""


constexpr int N = 1024;

static  __global__ void matrixMul(const int* a, const int* b, int* c, int N) {
	// Compute each thread's global row and column index
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// Iterate over row, and down column
	c[row * N + col] = 0;
	for (int k = 0; k < N; k++) {
		// Accumulate results for a single element
		c[row * N + col] += a[row * N + k] * b[k * N + col];
	}
}

void verify_result(int* a, int* b, int* c, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++) {
				c[i * N + j] += a[i * N + k] * b[k * N + j];
			}
		}
	}
}


int main() {
	int size = N * N * sizeof(int);

	// Allocate CPU Memory
	int* h_a = (int*)malloc(size);
	int* h_b = (int*)malloc(size);
	int* h_c = (int*)malloc(size);

	// Initialize Matrix
	for (int i = 0; i < N * N; i++) {
		h_a[i] = rand() % 100;  // Random integer between 0 and 99
		h_b[i] = rand() % 100;
	}

	// Initialize GPU Var
	int* d_a, * d_b, * d_c;

	// Allocate device memory
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	// Copy data to the device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	// Threads per CTA dimension
	int THREADS = 32;

	// Blocks per grid dimension (assumes THREADS divides N evenly)
	int BLOCKS = N / THREADS;

	// Use dim3 structs for block  and grid dimensions
	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	// Launch kernel
	matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);
	hipDeviceSynchronize();

	// Copy back to the host
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Check result
	verify_result(h_a, h_b, h_c, N);

	printf("COMPLETED SUCCESSFULLY\n");

	float GPU_time = 0.0f;
	hipEventElapsedTime(&GPU_time, start, stop);
	printf("%f ms \n",GPU_time);

	// Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	system("pause");
	return 0;
}